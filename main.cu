#include "hip/hip_runtime.h"
"""
__global__ indicates a CUDA kernel function that will be executed on the GPU.

helloWorld is the kernel function that prints 'Hello, World!'.

helloWorld<<<1, 1>>>(); launches the kernel with one block of one thread.

hipDeviceSynchronize(); ensures that the CPU waits for the GPU to finish before proceeding.
 
"""

#include <stdio.h>

__global__ void helloWorld() {
    printf("Hello, World!\n");
}

int main() {
    // Launch kernel on the default stream
    helloWorld<<<1, 1>>>();
    
    // Wait for the GPU to finish before accessing on host
    hipDeviceSynchronize();

    return 0;
}
